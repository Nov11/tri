#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <string>
#include <fstream>
#include <unordered_map>
#include <algorithm>
#include <iterator>
#include "Header.h"
#include "hip/hip_runtime_api.h"
#include "timeutility.h"
#include <>


//
//struct Intersector{
//	typedef thrust::device_vector<thrust::device_vector<int>> ADJLIST;
//	Intersector(ADJLIST& para, int pi) :adjl(para), i(pi){}
//	__device__
//	int operator()(int j){
//		const thrust::device_vector<int>& idev = adjl[i];
//		const thrust::device_vector<int>& jdev = adjl[j];
//		int ret = intersect(idev, jdev);
//		return ret;
//	}
//	typedef thrust::device_vector<int> DEVARRAY;
//	int intersect(const DEVARRAY& i, const DEVARRAY& j)
//	{
//		int ret = 0;
//		for (DEVARRAY::iterator iteri = i.begin(), iterj = j.begin(); iteri != i.end() && iterj != j.end();){
//			if (*iteri == *iterj){
//				ret++;
//				iteri++;
//				iterj++;
//			}
//			else if (*iteri < *iterj){
//				iteri++;
//			}
//			else{
//				iterj++;
//			}
//		}
//		return ret;
//	}
//	ADJLIST& adjl;
//	int i;
//};
//
//struct Op{
//	typedef thrust::device_vector<thrust::device_vector<int>> ADJLIST;
//	Op(ADJLIST& para) :adjl(para){}
//	__device__
//	int operator()(int i){
//		const thrust::device_vector<int>& dref = adjl[i];
//		thrust::device_vector<int> dev(dref.size());
//		thrust::transform(dref.begin(), dref.end(), dev.begin(), Intersector(adjl, i));
//		int ret = thrust::reduce(dev.begin(), dev.end());
//		return ret;
//	}
//	ADJLIST& adjl;
//};
//
//int c_forward(AdjList& adjl)
//{
//	int ret = 0;
//	thrust::device_vector<int> dev(adjl.node_num);
//	thrust::sequence(dev.begin(), dev.end());
//	thrust::device_vector<thrust::device_vector<int>> dev_adjl(adjl.adj_list);
//	//thrust::transform(dev.begin(), dev.end(), dev.begin(), Op(dev_adjl));
//	ret = thrust::reduce(dev.begin(), dev.end());
//	return ret;
//}
//
//int cuda_forward(std::ifstream& ifs)
//{
//	AdjList adjl;
//	{
//		OriEdgeList oel;
//		read_edges(ifs, oel);
//		make_adj_list_cuda(oel, adjl);
//	}
//	int ret = c_forward(adjl);
//	return ret;
//}




void copyEdgeListToDev(OriEdgeList& oel, int*& dev)
{
	hipStream_t stream0;
	checkCudaErrors(hipStreamCreate(&stream0));
	checkCudaErrors(hipMalloc((void**)&dev, oel.edge_num * 2 * sizeof(int) ));
	checkCudaErrors(hipMemcpyAsync(dev, oel.edges.data(), oel.edge_num * 2 * sizeof(int), hipMemcpyHostToDevice, stream0));
	checkCudaErrors(hipDeviceSynchronize());
}

void freeEdgeListFromDev(int*& dev)
{
	checkCudaErrors(hipFree(dev));
}

void copyAdjListToDev(AdjList& adjl, int*& ptr)
{
	hipStream_t stream0;
	checkCudaErrors(hipStreamCreate(&stream0));
	checkCudaErrors(hipMalloc((void**)&ptr, adjl.adj_list.size() * sizeof(int*) ));
	std::vector<int*> dev_ptr(adjl.adj_list.size());
	for (size_t i = 0; i < adjl.adj_list.size(); i++){
		checkCudaErrors(hipMalloc((void**)&dev_ptr[i], sizeof(int) * adjl.adj_list[i].size()));
		checkCudaErrors(hipMemcpyAsync(dev_ptr[i], adjl.adj_list[i].data(), sizeof(int) * adjl.adj_list[i].size(), hipMemcpyHostToDevice, stream0));
	}
	checkCudaErrors(hipMemcpyAsync(ptr, dev_ptr.data(), sizeof(int*) * adjl.adj_list.size(), hipMemcpyHostToDevice, stream0));
	checkCudaErrors(hipDeviceSynchronize());
}

int cuda_reverse_edge_foward(AdjList& adjl)
{
	int ret = 0;
	return ret;
}

typedef std::vector<int>::iterator Iter;
int intersect_iter(Iter n1beg, Iter n1end, Iter n2beg, Iter n2end)
{
	int ret = 0;
	while (n1beg != n1end && n2beg != n2end){
		if (*n1beg == *n2beg){
			ret++;
			n1beg++;
			n2beg++;
		}
		else if (*n1beg < *n2beg){
			n1beg++;
		}
		else {
			n2beg++;
		}
	}
	return ret;
}

int forward_no_explicit_edge_list(std::vector<int>& index, std::vector<int>& edge_head)
{
	int ret = 0;
	Iter beg = edge_head.begin();
	for (size_t i = 0; i < index.size() - 1; i++){
		for (Iter eh = beg + index[i]; eh != beg + index[i + 1]; eh++){
			int h = *eh;
			ret += intersect_iter(beg + index[i], beg + index[i + 1], beg + index[h], beg + index[h + 1]);
		}
	}
	return ret;
}

int forward_with_edge_list(std::vector<int>& index, std::vector<int>& edge_head, std::vector<int>& tail)
{
	int ret = 0;
	for (size_t i = 0; i < edge_head.size(); i++){
		int node1 = tail[i];
		int node2 = edge_head[i];
		Iter beg = edge_head.begin();
		ret += intersect_iter(beg + index[node1], beg + index[node1 + 1], beg + index[node2], beg + index[node2 + 1]);
	}
	return ret;
}

inline int do_serial_cuda_forward(std::vector<int>& edge_tail_start_index, std::vector<int>& edge_head, std::vector<int>& tail)
{
	CpuTime ct;
	ct.startTimer();
	int ret = forward_no_explicit_edge_list(edge_tail_start_index, edge_head);//cuda_reverse_edge_foward(adjl);
	//int ret = r_e_l_forward(edge_tail_start_index, edge_head, tail);
	ct.stopAndPrint("!!!reverse_edge_list_forward core!!!");
	return ret;
}

__device__ int insect(int* beg1, int* end1, int* beg2, int* end2)
{
	int ret = 0;
	while (beg1 != end1 && beg2 != end2){
		if (*beg1 == *beg2){
			ret++;
			beg1++;
			beg2++;
		}
		else if (*beg1 < *beg2){
			beg1++;
		}
		else{
			beg2++;
		}
	}
	return ret;
}

__global__ void kernel_counting(int* index, size_t index_len, int* head, size_t head_len, int* tail, int* result)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int step = gridDim.x * blockDim.x;
	int h, t;
	for (; idx < head_len; idx += step){
		h = head[idx];
		t = tail[idx];
		result[idx] += insect(head + index[h], head + index[h + 1], head + index[t], head + index[t + 1]);
	}
}

int cuda_count_triangle(int* dev_index, size_t index_len, int* dev_head, size_t head_len, int* dev_tail, int* dev_result)
{
	int ret = 0;
	int thread = 256;
	int block = (head_len - 1) / thread + 1;
	CpuTime ct;
	ct.startTimer();
	kernel_counting<<<block, thread>>>(dev_index, index_len, dev_head, head_len, dev_tail, dev_result);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	ret = thrust::reduce(thrust::device_ptr<int>(dev_result), thrust::device_ptr<int>(dev_result + head_len));
	ct.stopAndPrint("!!!cuda_count_triangle!!!");
	return ret;
}

int para_cuda_forward(std::vector<int>& index, std::vector<int>& head, std::vector<int>& tail)
{
	int ret = 0;
	int* dev_index;
	int* dev_head;
	int* dev_tail;
	int* result;

	hipStream_t stream1;
	hipStream_t stream2;
	CpuTime ct;
	ct.startTimer();
	checkCudaErrors(hipStreamCreate(&stream1));
	checkCudaErrors(hipStreamCreate(&stream2));
	checkCudaErrors(hipMalloc((void**)&result, sizeof(int) * (head.size()) ));
	checkCudaErrors(hipMemsetAsync(result, 0, sizeof(int) * (head.size()), stream1));

	checkCudaErrors(hipMalloc((void**)&dev_index, sizeof(int) * index.size()));
	checkCudaErrors(hipMemcpyAsync(dev_index, index.data(), sizeof(int) * index.size(), hipMemcpyHostToDevice, stream1));
	checkCudaErrors(hipMalloc((void**)&dev_head, sizeof(int) * head.size()));
	checkCudaErrors(hipMemcpyAsync(dev_head, head.data(), sizeof(int) * head.size(), hipMemcpyHostToDevice, stream2));
	checkCudaErrors(hipMalloc((void**)&dev_tail, sizeof(int) * tail.size()));
	checkCudaErrors(hipMemcpyAsync(dev_tail, tail.data(), sizeof(int) * tail.size(), hipMemcpyHostToDevice, stream2));

	checkCudaErrors(hipDeviceSynchronize());
	ct.stopAndPrint("@@@copy to device done");

	ret = cuda_count_triangle(dev_index, index.size(), dev_head, head.size(), dev_tail, result);
	return ret;
}

void preprocess(std::ifstream& ifs, std::vector<int>& edge_tail_start_index, std::vector<int>& edge_head)
{
	AdjList adjl;
	{
		OriEdgeList oel;
		read_edges(ifs, oel);
		make_adj_list_cuda(oel, adjl);
		//copyEdgeListToDev(oel, dev_edge_list);
		//copyAdjListToDev(adjl, dev_adj_list);//4.39s in kernel 7.969s in userspace for amazon ungraph, it's too slow
		//std::cout << "on device " << dev[0] << " " << dev[1] << std::endl;
		//std::cout << "on host " << oel.edges[0].first << " " << oel.edges[0].second << std::endl;
		//thrust::device_vector<thrust::device_vector<int>> dev_vec(adjl.adj_list.size());
		//for (size_t i = 0; i < adjl.adj_list.size(); i++){
		//	dev_vec[i] = adjl.adj_list[i];
		//}
		//std::cout << "dev_vec size: " << dev_vec.size() << std::endl;
	}
	int edge_count = 0;
	edge_tail_start_index.resize(adjl.adj_list.size() + 1);
	edge_head.reserve(adjl.edge_num);
	for (size_t i = 0; i < adjl.adj_list.size(); i++){
		edge_tail_start_index[i] = edge_count;
		edge_count += adjl.adj_list[i].size();
		std::copy(adjl.adj_list[i].begin(), adjl.adj_list[i].end(), std::back_inserter(edge_head));
	}
	edge_tail_start_index.back() = edge_count;
}

int cuda_forward(std::ifstream& ifs)
{
	
	std::vector<int> edge_tail_start_index;
	std::vector<int> edge_head;
	preprocess(ifs, edge_tail_start_index, edge_head);

	std::vector<int> tail;
	tail.resize(edge_head.size());
	for (size_t i = 0; i < edge_tail_start_index.size() - 1; i++){
		std::fill(tail.begin() + edge_tail_start_index[i], tail.begin() + edge_tail_start_index[i + 1], i);
	}

	int ret = para_cuda_forward(edge_tail_start_index, edge_head, tail);

	checkCudaErrors(hipDeviceReset());
	return ret;
}

int serial_cuda_forward_without_tail(std::ifstream& ifs)
{
	std::vector<int> edge_tail_start_index;
	std::vector<int> edge_head;
	preprocess(ifs, edge_tail_start_index, edge_head);

	CpuTime ct;
	ct.startTimer();
	int ret = forward_no_explicit_edge_list(edge_tail_start_index, edge_head);
	ct.stopAndPrint("!!!serial cuda forward without tail!!!");
	return ret;
}

int serial_cuda_forward(std::ifstream& ifs)
{
	std::vector<int> edge_tail_start_index;
	std::vector<int> edge_head;
	preprocess(ifs, edge_tail_start_index, edge_head);

	std::vector<int> tail;
	tail.resize(edge_head.size());
	for (size_t i = 0; i < edge_tail_start_index.size() - 1; i++){
		std::fill(tail.begin() + edge_tail_start_index[i], tail.begin() + edge_tail_start_index[i + 1], i);
	}
	CpuTime ct;
	ct.startTimer();
	int ret = forward_with_edge_list(edge_tail_start_index, edge_head, tail);
	ct.stopAndPrint("!!!serial cuda forward with tail!!!");
	return ret;
}
//void cuda_forward(std::string& file, int result[], int item)
//{
//	std::ifstream ifs(file);
//	if (!ifs){
//		std::cerr << "error open file" << std::endl;
//		exit(1);
//	}
//	OriEdgeList oel;
//	read_edges(ifs, oel);
//	thrust::device_vector<std::pair<int, int>> dev_vec(oel.edges.begin(), oel.edges.end());
//}
